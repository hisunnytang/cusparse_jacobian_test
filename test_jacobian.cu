#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#include <sundials/sundials_types.h>
#include <sunlinsol/sunlinsol_cusolversp_batchqr.h>
#include <sunmatrix/sunmatrix_dense.h>
#include <sunmatrix/sunmatrix_sparse.h>
#include <sunmatrix/sunmatrix_cusparse.h>
#include <nvector/nvector_cuda.h>
#include <nvector/nvector_serial.h>
#include <sundials/sundials_math.h>
#include "test_sunlinsol.h"

#define nchem 5
#define nnz 25
#define batchsize 4
#define gridsize 2
#define blocksize batchsize/gridsize

static int check_retval(void *returnvalue, const char *funcname, int opt)
{
  int *retval;

  /* Check if SUNDIALS function returned NULL pointer - no memory allocated */
  if (opt == 0 && returnvalue == NULL) {
    fprintf(stderr, "\nSUNDIALS_ERROR: %s() failed - returned NULL pointer\n\n",
	    funcname);
    return(1); }

  /* Check if retval < 0 */
  else if (opt == 1) {
    retval = (int *) returnvalue;
    if (*retval < 0) {
      fprintf(stderr, "\nSUNDIALS_ERROR: %s() failed with retval = %d\n\n",
	      funcname, *retval);
      return(1); }}

  /* Check if function returned NULL pointer - no memory allocated */
  else if (opt == 2 && returnvalue == NULL) {
    fprintf(stderr, "\nMEMORY_ERROR: %s() failed - returned NULL pointer\n\n",
	    funcname);
    return(1); }

  return(0);
}
static int blockJacInit(SUNMatrix J)
{
    
    int rowptrs[nchem+1];
    int colvals[nnz];

    SUNMatZero(J);
    for (int r = 0; r < nchem+1; r++)
    {
        rowptrs[r] = r*nchem;
        printf("rowptrs[%d] = %d\n", r, rowptrs[r]);
    }

    int bIdx;
    for (int c = 0; c < nnz; c++)
    {
        bIdx = c /nnz; 
        colvals[c] = bIdx*nchem + c%nchem;
        printf("colvals[%d] = %d\n", c, colvals[c]);
    }
    // copy rowptrs, colvals to the device
    SUNMatrix_cuSparse_CopyToDevice(J, NULL, rowptrs, colvals);
    hipDeviceSynchronize();
    return (0);
}


static int JacInit(SUNMatrix J)
{
    
    int rowptrs[batchsize*nchem+1];
    int colvals[batchsize*nnz  ];

    SUNMatZero(J);

    for (int r = 0; r < batchsize*nchem+1; r++)
    {
        rowptrs[r] = r*nchem;
        printf("rowptrs[%d] = %d\n", r, rowptrs[r]);
    }

    int bIdx;
    for (int c = 0; c < batchsize*nnz; c++)
    {
        bIdx = c /nnz; 
        colvals[c] = bIdx*nchem + c%nchem;
        printf("colvals[%d] = %d\n", c, colvals[c]);
    }
    // copy rowptrs, colvals to the device
    SUNMatrix_cuSparse_CopyToDevice(J, NULL, rowptrs, colvals);
    hipDeviceSynchronize();
    return (0);
}

__global__
static void jacobian_kernel(realtype *Jdata)
{
    int groupj = blockIdx.x*blockDim.x + threadIdx.x; 
    if (groupj < batchsize)
    {
        for (int i = 0; i < nnz; i++)
        {
            if (i%2 == 0)
            {
                Jdata[groupj*nnz+i] = 1;
            }else{
                Jdata[groupj*nnz+i] = 0;
            }
        }
    }
}

static int Jacobian(SUNMatrix J)
{
    realtype *Jdata;
    Jdata = SUNMatrix_cuSparse_Data(J);
    jacobian_kernel<<<gridsize, blocksize>>>(Jdata);
    
    hipDeviceSynchronize();
    hipError_t cuerr = hipGetLastError();
    if (cuerr != hipSuccess) {
        fprintf(stderr, ">>> ERROR in Jac: hipGetLastError returned %s\n",
                hipGetErrorName(cuerr));
        return(-1);
    }

  return(0);
}


int main()
{

  SUNLinearSolver LS;                 /* linear solver object          */
  hipsparseStatus_t cusp_status;
  hipsolverStatus_t cusol_status;
  hipsparseHandle_t cusp_handle;
  hipsolverSpHandle_t cusol_handle;
  /* Initialize cuSPARSE */
  cusp_status = hipsparseCreate(&cusp_handle);
  if (cusp_status != HIPSPARSE_STATUS_SUCCESS) {
    printf("ERROR: could not create cuSPARSE handle\n");
    return(-1);
  }

  /* Initialize cuSOLVER */
  cusol_status = hipsolverSpCreate(&cusol_handle);
  if (cusol_status != HIPSOLVER_STATUS_SUCCESS) {
    printf("ERROR: could not create cuSOLVER handle\n");
    return(-1);
  }


  int N = nchem*batchsize;
  N_Vector d_x, d_b;
  d_x = N_VNew_Cuda(N);
  d_b = N_VNew_Cuda(N);

  realtype *xdata, *bdata;
  xdata = N_VGetHostArrayPointer_Cuda(d_x);
  bdata = N_VGetHostArrayPointer_Cuda(d_b);
  for (int i=0; i<N; i++)
  {
      xdata[i] = i % 2;
      bdata[i] = 1.0 + i%2;
  }
  N_VCopyToDevice_Cuda(d_x);
  N_VCopyToDevice_Cuda(d_b);


  /* Create the device matrix */
  SUNMatrix J;
  //J = SUNMatrix_cuSparse_NewCSR(N, N, N*nchem, cusp_handle);
  // JacInit(J);
  // Jacobian(J);
  // Instead of using the CSR, we use BCSR
  J = SUNMatrix_cuSparse_NewBlockCSR(batchsize, nchem, nchem, nchem*nchem, cusp_handle);
  if(check_retval((void *)J, "SUNMatrix_cuSparse_NewBlockCSR", 0)) return(1);
  SUNMatrix_cuSparse_SetFixedPattern(J, 1);
  JacInit(J);
  Jacobian(J);


  /*
  // get the rows and col in the Sparse matrix
  int M = SUNMatrix_cuSparse_Rows(J);
  int N = SUNMatrix_cuPsarse_Columns(J);
  int nz = SUNMatrix_cuSparse_NNZ(J);
*/


  // create an empty host array to store it and print it
  SUNMatrix Jhost;
  Jhost = SUNSparseMatrix(N,N, N*nchem, CSR_MAT);

    SUNMatrix_cuSparse_CopyFromDevice(J,
            SUNSparseMatrix_Data(Jhost),
            SUNSparseMatrix_IndexPointers(Jhost),
            SUNSparseMatrix_IndexValues(Jhost));

    printf("\nJhost =\n");
    SUNSparseMatrix_Print(Jhost,stdout);

    // create a linear solver object
    // from J


    LS = SUNLinSol_cuSolverSp_batchQR(d_x, J, cusol_handle);


  if (LS == NULL) {
    printf("FAIL: SUNLinSol_cuSolverSp_batchQR returned NULL\n");
    return(1);
  }

  // need to first initialize sunlinsol
  SUNLinSolInitialize(LS);
  sync_device();
  // first we need a linsolsetup;
  int failure;
  failure = SUNLinSolSetup(LS, J);
  sync_device();

  N_Vector tmp;
  tmp = N_VClone(d_x);

  // perform solve
  failure = SUNLinSolSolve(LS, J, d_x, d_b, 0.001);
  sync_device();


    N_VCopyFromDevice_Cuda(d_x); /* copy solution from device */
    printf("x (computed)\n");
    N_VPrint_Cuda(d_x);

    N_VCopyFromDevice_Cuda(d_b);
    printf("\nb = Ax (reference)\n");
    N_VPrint_Cuda(d_b);

}


/* ----------------------------------------------------------------------
 * Implementation-specific 'check' routines
 * --------------------------------------------------------------------*/
int check_vector(N_Vector X, N_Vector Y, realtype tol)
{
  int failure = 0;
  sunindextype i, local_length, maxloc;
  realtype *Xdata, *Ydata, maxerr;

  hipDeviceSynchronize();

  N_VCopyFromDevice_Cuda(X);
  N_VCopyFromDevice_Cuda(Y);

  Xdata = N_VGetHostArrayPointer_Cuda(X);
  Ydata = N_VGetHostArrayPointer_Cuda(Y);
  local_length = N_VGetLength(X);

  /* check vector data */
  for(i=0; i < local_length; i++)
    failure += FNEQ(Xdata[i], Ydata[i], tol);

  if (failure > ZERO) {
    maxerr = ZERO;
    maxloc = -1;
    for(i=0; i < local_length; i++) {
      if (SUNRabs(Xdata[i]-Ydata[i]) >  maxerr) {
        maxerr = SUNRabs(Xdata[i]-Ydata[i]);
        maxloc = i;
      }
    }
    printf("check err failure: maxerr = %g at loc %li (tol = %g)\n",
	   maxerr, (long int) maxloc, tol);
    return(1);
  }
  else
    return(0);
}

void sync_device()
{
  hipDeviceSynchronize();
}
