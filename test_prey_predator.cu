#include "hip/hip_runtime.h"
#include "hdf5.h"
#include "hdf5_hl.h"
#include <stdio.h>
#include <cvode/cvode.h>                  /* prototypes for CVODE fcts., consts.          */
#include <nvector/nvector_cuda.h>         /* access to cuda N_Vector                      */
#include <sunmatrix/sunmatrix_cusparse.h>             /* access to cusparse SUNMatrix                  */
#include <sunlinsol/sunlinsol_cusolversp_batchqr.h>   /* acess to cuSolverSp batch QR SUNLinearSolver */
#include <sundials/sundials_types.h>     /* defs. of realtype, int              */
#include "test_sunlinsol.h"
#include <sundials/sundials_math.h>
#define BATCHSIZE 1024
#define ZERO    RCONST(0.0)
#define kb      RCONST(1.3806504e-16)
#define mh      RCONST(1.67e-24)
#define gamma   RCONST(5.0/3.0)
#define _gamma_m1 RCONST(1.0/ (gamma-1.0) )

#define GRIDSIZE 32
#define BLOCKSIZE 32

#define T0 RCONST(0.0)
#define T1 RCONST(1e10)
#define TMULT RCONST(2.0)
#define NOUT 10 
// define our datatype
typedef struct
{
    double nbins;
    double dbin;
    double idbin;
    double lb;
    double ub;

    double current_z;
    double *Ts;
    double *logTs;
    double *Tdef;
    double *dTs_ge;
    double *Tge;

    // cooling & chemical tables
    double *r_exp_growth_prey;
    double *rs_exp_growth_prey;
    double *drs_exp_growth_prey;
    double *r_natural_death_predator;
    double *rs_natural_death_predator;
    double *drs_natural_death_predator;
    double *r_predation;
    double *rs_predation;
    double *drs_predation;

    // for now, we ignore the temperature dependent Gamma
    /*
    double *g_gamma;
    double *g_dgamma_dT;
    double *gamma;
    double *dgamma_dT;
    double *_gamma_dT;
    double *g_gamma;
    double *g_dgamma_dT;
    double *gamma;
    double *dgamma_dT;
    double *_gamma_dT;
    */


} abc_data;


// Initialize a data object that stores the reaction/ cooling rate data
abc_data abc_setup_data(int *NumberOfFields, char ***FieldNames)
{

    //-----------------------------------------------------
    // Function : abc_setup_data
    // Description: Initialize a data object that stores the reaction/ cooling rate data 
    //-----------------------------------------------------


    // let's not re-scale the data yet...
    abc_data ratedata;

    ratedata.nbins = 1024;
    ratedata.dbin = (log( 100000000.0)-log(1.0)) / 1023;
    ratedata.idbin = 1.0 / ratedata.dbin;
    ratedata.lb   = log(1.0);
    ratedata.ub   = log(100000000.0);

    /* Redshift-related pieces */
    /*
    data->z_bounds[0] = 0.0;
    data->z_bounds[1] = 10.0;
    data->n_zbins = 0 - 1;
    data->d_zbin = (log(data->z_bounds[1] + 1.0) - log(data->z_bounds[0] + 1.0)) / data->n_zbins;
    data->id_zbin = 1.0L / data->d_zbin;
    */

    
    // initialize memory space for reaction rates and cooling rates
    // we use managed data, so the pointer can simultaneously be accessed from device and the host
    hipMallocManaged(&ratedata.r_exp_growth_prey, sizeof(double)*1024);
    hipMallocManaged(&ratedata.rs_exp_growth_prey, sizeof(double)*1024);
    hipMallocManaged(&ratedata.drs_exp_growth_prey, sizeof(double)*1024);
    hipMallocManaged(&ratedata.r_natural_death_predator, sizeof(double)*1024);
    hipMallocManaged(&ratedata.rs_natural_death_predator, sizeof(double)*1024);
    hipMallocManaged(&ratedata.drs_natural_death_predator, sizeof(double)*1024);
    hipMallocManaged(&ratedata.r_predation, sizeof(double)*1024);
    hipMallocManaged(&ratedata.rs_predation, sizeof(double)*1024);
    hipMallocManaged(&ratedata.drs_predation, sizeof(double)*1024);

    // Cooling Rates

    // initialize memory space for the temperature-related pieces
    hipMallocManaged(&ratedata.Ts, sizeof(double)* BATCHSIZE);
    hipMallocManaged(&ratedata.logTs, sizeof(double)* BATCHSIZE);
    hipMallocManaged(&ratedata.Tdef,  sizeof(double)* BATCHSIZE);
    hipMallocManaged(&ratedata.dTs_ge,  sizeof(double)* BATCHSIZE);
    hipMallocManaged(&ratedata.Tge,  sizeof(double)* BATCHSIZE);

    // gamma as a function of temperature
    /*
    hipMallocManaged(&ratedata.g_gammaH2_1, sizeof(double)* BATCHSIZE);
    hipMallocManaged(&ratedata.g_dgammaH2_1_dT,  sizeof(double)* BATCHSIZE);
    hipMallocManaged(&ratedata.gammaH2_1,  sizeof(double)* BATCHSIZE);
    hipMallocManaged(&ratedata.dgamma_dTH2_1,  sizeof(double)* BATCHSIZE);
    hipMallocManaged(&ratedata._gammaH2_1_dT, sizeof(double)*BATCHSIZE);
    hipMallocManaged(&ratedata.g_gammaH2_2, sizeof(double)* BATCHSIZE);
    hipMallocManaged(&ratedata.g_dgammaH2_2_dT,  sizeof(double)* BATCHSIZE);
    hipMallocManaged(&ratedata.gammaH2_2,  sizeof(double)* BATCHSIZE);
    hipMallocManaged(&ratedata.dgamma_dTH2_2,  sizeof(double)* BATCHSIZE);
    hipMallocManaged(&ratedata._gammaH2_2_dT, sizeof(double)*BATCHSIZE);
    
    // maybe we can calculate the density on the fly
    // space to store the mass density
    hipMallocManaged(&ratedata.mdensity, sizeof(double)* BATCHSIZE);
    hipMallocManaged(&ratedata.inv_mdensity, sizeof(double)* BATCHSIZE);
    */
    // extra stuff like the density-dependent cooling rate
    
    return ratedata;
}


void abc_read_rate_tables(abc_data *data)
{
    hid_t file_id = H5Fopen("abc_tables.h5", H5F_ACC_RDONLY, H5P_DEFAULT);
    /* Allocate the correct number of rate tables */
    H5LTread_dataset_double(file_id, "/exp_growth_prey", data->r_exp_growth_prey);
    H5LTread_dataset_double(file_id, "/natural_death_predator", data->r_natural_death_predator);
    H5LTread_dataset_double(file_id, "/predation", data->r_predation);
    
    H5Fclose(file_id);
}


void abc_read_cooling_tables(abc_data *data)
{

    hid_t file_id = H5Fopen("abc_tables.h5", H5F_ACC_RDONLY, H5P_DEFAULT);
    /* Allocate the correct number of rate tables */

    H5Fclose(file_id);
}

/*
void abc_read_gamma(abc_data *data)
{

    hid_t file_id = H5Fopen("abc_tables.h5", H5F_ACC_RDONLY, H5P_DEFAULT);
    // Allocate the correct number of rate tables
    H5LTread_dataset_double(file_id, "/gammaH2_1",
                            data->g_gammaH2_1 );
    H5LTread_dataset_double(file_id, "/dgammaH2_1_dT",
                            data->g_dgammaH2_1_dT );   
    
    H5LTread_dataset_double(file_id, "/gammaH2_2",
                            data->g_gammaH2_2 );
    H5LTread_dataset_double(file_id, "/dgammaH2_2_dT",
                            data->g_dgammaH2_2_dT );   
    

    H5Fclose(file_id);

}
 
*/


// interpolation kernel
// ideally, we should use texture to do interpolation,
// let's ignore it for now, cos i guess most time is spent in doing the matrix thingy

__global__
void linear_interpolation_kernel(abc_data data)
{
    int j = threadIdx.x + blockDim.x* blockIdx.x;
    
    int k;
    double Tdef, t1;
    double *exp_growth_prey = data.r_exp_growth_prey;
    double *rs_exp_growth_prey  = data.rs_exp_growth_prey;
    double *drs_exp_growth_prey = data.drs_exp_growth_prey;
    double *natural_death_predator = data.r_natural_death_predator;
    double *rs_natural_death_predator  = data.rs_natural_death_predator;
    double *drs_natural_death_predator = data.drs_natural_death_predator;
    double *predation = data.r_predation;
    double *rs_predation  = data.rs_predation;
    double *drs_predation = data.drs_predation;
    
    if (j < BATCHSIZE)
    {
        k    = __float2int_rz(data.idbin*data.logTs[j] - data.lb);
        t1   = data.lb + k*data.dbin;
        Tdef = (data.logTs[j] - t1) * data.idbin;
        rs_exp_growth_prey[j] = Tdef*exp_growth_prey[k+1] + (-exp_growth_prey[k]*Tdef + exp_growth_prey[k]);
        rs_natural_death_predator[j] = Tdef*natural_death_predator[k+1] + (-natural_death_predator[k]*Tdef + natural_death_predator[k]);
        rs_predation[j] = Tdef*predation[k+1] + (-predation[k]*Tdef + predation[k]);
        
    }
}


__global__
static void rhs_kernel(double y, double *ydata, double *ydotdata, abc_data data)
{
    int i = blockIdx.x* blockDim.x + threadIdx.x;

    int nchem  = 5;
    int groupi = i * nchem; 

    // get rate pointer
    double *exp_growth_prey = data.rs_exp_growth_prey;
    double *natural_death_predator = data.rs_natural_death_predator;
    double *predation = data.rs_predation;

    
    
    

    int j;
    double z, T, mdensity, inv_mdensity;

    if (i < BATCHSIZE)
    {
        T = data.Ts[i];
        z = data.current_z;

        
        
        double dead_predator = ydata[groupi+0];
        double dead_prey = ydata[groupi+1];
        double ge = ydata[groupi+2];
        double predator = ydata[groupi+3];
        double prey = ydata[groupi+4];

        mdensity     = mh*(1.0*dead_predator + 1.0*dead_prey + 1.0*predator + 1.0*prey);
        inv_mdensity = 1.0/mdensity;
        //
        // Species: dead_predator
        //
        j = 0;
        ydotdata[groupi+j] = natural_death_predator[i]*predator;
        
        j++;
        //
        // Species: dead_prey
        //
        j = 1;
        ydotdata[groupi+j] = predation[i]*predator*prey;
        
        j++;
        //
        // Species: ge
        //
        j = 2;
        ydotdata[groupi+j] = 0;
        
        ydotdata[groupi+j] *= inv_mdensity;
        
        j++;
        //
        // Species: predator
        //
        j = 3;
        ydotdata[groupi+j] = -natural_death_predator[i]*predator + 0.75*predation[i]*predator*prey;
        
        j++;
        //
        // Species: prey
        //
        j = 4;
        ydotdata[groupi+j] = exp_growth_prey[i]*prey - predation[i]*predator*prey;
        
        j++;

    }
    
    /*
    for (int ii = 0; ii < 5; ii++)
    {
        printf("from %d: ydot[%d] = %0.5g; ydata = %0.5g\n", i, ii, ydotdata[groupi+ii], ydata[groupi+ii]);
    }
    */

}


__global__
void temperature_kernel(double* ydata, abc_data data)
{
    int i = blockIdx.x* blockDim.x + threadIdx.x;
    int nchem  = 5;
    int groupi = i * nchem; 

    double *temperature = data.Ts;
    double *logTs      = data.logTs;
    double *Tge        = data.Tge;

    double gammaH2_1 = 7./5.;
    double gammaH2_2 = 7./5.;
    // as of now just do not use any "temperature-dependent" gamma
    // which simplifies my life, and not having the need to iterate it to convergence

    if (i < BATCHSIZE)
    {
        double dead_predator = ydata[groupi+0];
        double dead_prey = ydata[groupi+1];
        double ge = ydata[groupi+2];
        double predator = ydata[groupi+3];
        double prey = ydata[groupi+4];
        double density = 1.0*dead_predator + 1.0*dead_prey + 1.0*predator + 1.0*prey;
        temperature[i] = 100; //density*ge*mh/(kb*(_gamma_m1*dead_predator + _gamma_m1*dead_prey + _gamma_m1*predator + _gamma_m1*prey));
        logTs      [i] = log(temperature[i]);
        Tge        [i] = 0.0; //TODO: update this to dT_dge;
    }
}

// Function Called by the solver
static int f(realtype t, N_Vector y, N_Vector ydot, void *user_data)
{
    abc_data *udata = (abc_data *) user_data;
    double *ydata    = N_VGetDeviceArrayPointer_Cuda(y);
    double *ydotdata = N_VGetDeviceArrayPointer_Cuda(ydot);

    // calculate temperature kernel
    temperature_kernel<<<GRIDSIZE, BLOCKSIZE>>> (ydata, *udata);
    // interpolate the rates with updated temperature
    linear_interpolation_kernel<<<GRIDSIZE, BLOCKSIZE>>>(*udata);

    // update ydot with the kernel function
    rhs_kernel<<<GRIDSIZE, BLOCKSIZE>>>(t, ydata, ydotdata, *udata);

    hipDeviceSynchronize();
    hipError_t cuerr = hipGetLastError();
    if (cuerr != hipSuccess) {
        fprintf(stderr,
        ">>> ERROR in f: hipGetLastError returned %s\n",
        hipGetErrorName(cuerr));
        return(-1);
    }


    return 0;

}


// write jacobian



/*
 * taken from cvRoberts_block_cusolversp_batchqr.cu
 *
 * Jacobian initialization routine. This sets the sparisty pattern of
 * the blocks of the Jacobian J(t,y) = df/dy. This is performed on the CPU,
 * and only occurs at the beginning of the simulation.
 */
static int blockJacInit(SUNMatrix J)
{
    int nchem = 5;
    int nnz = 5*5;
    
    int rowptrs[nchem+1];
    int colvals[nnz];

    SUNMatZero(J);
    for (int r = 0; r < nchem+1; r++)
    {
        rowptrs[r] = r*nchem;
        printf("rowptrs[%d] = %d\n", r, rowptrs[r]);
    }

    int bIdx;
    for (int c = 0; c < nnz; c++)
    {
        bIdx = c /nnz; 
        colvals[c] = bIdx*nchem + c%nchem;
        printf("colvals[%d] = %d\n", c, colvals[c]);
    }
    // copy rowptrs, colvals to the device
    SUNMatrix_cuSparse_CopyToDevice(J, NULL, rowptrs, colvals);
    hipDeviceSynchronize();
    return (0);
}
static int JacInit(SUNMatrix J)
{
    int rowptrs[5+1];
    int colvals[5 * 5];

    /* Zero out the Jacobian */
    SUNMatZero(J);

    /* there are 5 entries per row*/
    rowptrs[0] = 0 * 5;
    rowptrs[1] = 1 * 5;
    rowptrs[2] = 2 * 5;
    rowptrs[3] = 3 * 5;
    rowptrs[4] = 4 * 5;
    
    // 0 row of block
    colvals[0] = 0;
    colvals[1] = 1;
    colvals[2] = 2;
    colvals[3] = 3;
    colvals[4] = 4;
    
    // 1 row of block
    colvals[5] = 0;
    colvals[6] = 1;
    colvals[7] = 2;
    colvals[8] = 3;
    colvals[9] = 4;
    
    // 2 row of block
    colvals[10] = 0;
    colvals[11] = 1;
    colvals[12] = 2;
    colvals[13] = 3;
    colvals[14] = 4;
    
    // 3 row of block
    colvals[15] = 0;
    colvals[16] = 1;
    colvals[17] = 2;
    colvals[18] = 3;
    colvals[19] = 4;
    
    // 4 row of block
    colvals[20] = 0;
    colvals[21] = 1;
    colvals[22] = 2;
    colvals[23] = 3;
    colvals[24] = 4;

    // copy rowptrs, colvals to the device
    SUNMatrix_cuSparse_CopyToDevice(J, NULL, rowptrs, colvals);
    hipDeviceSynchronize();
    return (0);
}


/* Jacobian evaluation with GPU */
__global__
static void jacobian_kernel(realtype *ydata, realtype *Jdata, abc_data data)
{
    int groupj;
    int GROUPSIZE = 5;
    
    // temporary:
    int nnzper = GROUPSIZE*GROUPSIZE;
    int i;
    double *Tge = data.Tge;
    double z, T;

    
    
    groupj = blockIdx.x*blockDim.x + threadIdx.x; 
    
    T = 1000.0;
    z = 0.0;

    if (groupj < BATCHSIZE)
    {
        i = groupj;
        
        
        // pulled the species data
        double dead_predator = ydata[GROUPSIZE*groupj+0];
        double dead_prey = ydata[GROUPSIZE*groupj+1];
        double ge = ydata[GROUPSIZE*groupj+2];
        double predator = ydata[GROUPSIZE*groupj+3];
        double prey = ydata[GROUPSIZE*groupj+4];
        double mdensity = mh * (1.0*dead_predator + 1.0*dead_prey + 1.0*predator + 1.0*prey);
        double inv_mdensity = 1.0/ mdensity;
        double *exp_growth_prey = data.rs_exp_growth_prey;
        double *rexp_growth_prey= data.drs_exp_growth_prey;
        double *natural_death_predator = data.rs_natural_death_predator;
        double *rnatural_death_predator= data.drs_natural_death_predator;
        double *predation = data.rs_predation;
        double *rpredation= data.drs_predation;
        //
        // Species: dead_predator
        //
        
        
        // dead_predator by dead_predator
        
        
        // because the Jacobian is initialized to zeros by default
        Jdata[nnzper*groupj+ 0*GROUPSIZE + 0] = ZERO;
        
        
        
 	    
        
        
        // dead_predator by dead_prey
        
        
        // because the Jacobian is initialized to zeros by default
        Jdata[nnzper*groupj+ 0*GROUPSIZE + 1] = ZERO;
        
        
        
 	    
        
        
        // dead_predator by ge
        
        
        Jdata[nnzper*groupj + 0*GROUPSIZE + 2] = predator*rnatural_death_predator[i];
        
        
        
 	    
        
        Jdata[nnzper*groupj+ 0*GROUPSIZE + 2] *= Tge[i];
        
        
        // dead_predator by predator
        
        
        Jdata[nnzper*groupj + 0*GROUPSIZE + 3] = natural_death_predator[i];
        
        
        
 	    
        
        
        // dead_predator by prey
        
        
        // because the Jacobian is initialized to zeros by default
        Jdata[nnzper*groupj+ 0*GROUPSIZE + 4] = ZERO;
        
        
        
 	    
        
        //
        // Species: dead_prey
        //
        
        
        // dead_prey by dead_predator
        
        
        // because the Jacobian is initialized to zeros by default
        Jdata[nnzper*groupj+ 1*GROUPSIZE + 0] = ZERO;
        
        
        
 	    
        
        
        // dead_prey by dead_prey
        
        
        // because the Jacobian is initialized to zeros by default
        Jdata[nnzper*groupj+ 1*GROUPSIZE + 1] = ZERO;
        
        
        
 	    
        
        
        // dead_prey by ge
        
        
        Jdata[nnzper*groupj + 1*GROUPSIZE + 2] = predator*prey*rpredation[i];
        
        
        
 	    
        
        Jdata[nnzper*groupj+ 1*GROUPSIZE + 2] *= Tge[i];
        
        
        // dead_prey by predator
        
        
        Jdata[nnzper*groupj + 1*GROUPSIZE + 3] = predation[i]*prey;
        
        
        
 	    
        
        
        // dead_prey by prey
        
        
        Jdata[nnzper*groupj + 1*GROUPSIZE + 4] = predation[i]*predator;
        
        
        
 	    
        
        //
        // Species: ge
        //
        
        
        // ge by dead_predator
        
        
        // because the Jacobian is initialized to zeros by default
        Jdata[nnzper*groupj+ 2*GROUPSIZE + 0] = ZERO;
        
        
        
        Jdata[nnzper*groupj+ 2*GROUPSIZE + 0] *= inv_mdensity;
        
 	    
        
        
        // ge by dead_prey
        
        
        // because the Jacobian is initialized to zeros by default
        Jdata[nnzper*groupj+ 2*GROUPSIZE + 1] = ZERO;
        
        
        
        Jdata[nnzper*groupj+ 2*GROUPSIZE + 1] *= inv_mdensity;
        
 	    
        
        
        // ge by ge
        
        
        // because the Jacobian is initialized to zeros by default
        Jdata[nnzper*groupj+ 2*GROUPSIZE + 2] = ZERO;
        
        
        
        Jdata[nnzper*groupj+ 2*GROUPSIZE + 2] *= inv_mdensity;
        
 	    
        
        Jdata[nnzper*groupj+ 2*GROUPSIZE + 2] *= Tge[i];
        
        
        // ge by predator
        
        
        // because the Jacobian is initialized to zeros by default
        Jdata[nnzper*groupj+ 2*GROUPSIZE + 3] = ZERO;
        
        
        
        Jdata[nnzper*groupj+ 2*GROUPSIZE + 3] *= inv_mdensity;
        
 	    
        
        
        // ge by prey
        
        
        // because the Jacobian is initialized to zeros by default
        Jdata[nnzper*groupj+ 2*GROUPSIZE + 4] = ZERO;
        
        
        
        Jdata[nnzper*groupj+ 2*GROUPSIZE + 4] *= inv_mdensity;
        
 	    
        
        //
        // Species: predator
        //
        
        
        // predator by dead_predator
        
        
        // because the Jacobian is initialized to zeros by default
        Jdata[nnzper*groupj+ 3*GROUPSIZE + 0] = ZERO;
        
        
        
 	    
        
        
        // predator by dead_prey
        
        
        // because the Jacobian is initialized to zeros by default
        Jdata[nnzper*groupj+ 3*GROUPSIZE + 1] = ZERO;
        
        
        
 	    
        
        
        // predator by ge
        
        
        Jdata[nnzper*groupj + 3*GROUPSIZE + 2] = 0.75*predator*prey*rpredation[i] - predator*rnatural_death_predator[i];
        
        
        
 	    
        
        Jdata[nnzper*groupj+ 3*GROUPSIZE + 2] *= Tge[i];
        
        
        // predator by predator
        
        
        Jdata[nnzper*groupj + 3*GROUPSIZE + 3] = -natural_death_predator[i] + 0.75*predation[i]*prey;
        
        
        
 	    
        
        
        // predator by prey
        
        
        Jdata[nnzper*groupj + 3*GROUPSIZE + 4] = 0.75*predation[i]*predator;
        
        
        
 	    
        
        //
        // Species: prey
        //
        
        
        // prey by dead_predator
        
        
        // because the Jacobian is initialized to zeros by default
        Jdata[nnzper*groupj+ 4*GROUPSIZE + 0] = ZERO;
        
        
        
 	    
        
        
        // prey by dead_prey
        
        
        // because the Jacobian is initialized to zeros by default
        Jdata[nnzper*groupj+ 4*GROUPSIZE + 1] = ZERO;
        
        
        
 	    
        
        
        // prey by ge
        
        
        Jdata[nnzper*groupj + 4*GROUPSIZE + 2] = -predator*prey*rpredation[i] + prey*rexp_growth_prey[i];
        
        
        
 	    
        
        Jdata[nnzper*groupj+ 4*GROUPSIZE + 2] *= Tge[i];
        
        
        // prey by predator
        
        
        Jdata[nnzper*groupj + 4*GROUPSIZE + 3] = -predation[i]*prey;
        
        
        
 	    
        
        
        // prey by prey
        
        
        Jdata[nnzper*groupj + 4*GROUPSIZE + 4] = exp_growth_prey[i] - predation[i]*predator;
        
        
        
 	    
        

    }

    /*
  if (groupj < 1){
      for (int i =0; i < 25; i++){
       printf("from %d: Jdata[%d] = %0.5g\n", groupj, i, Jdata[nnzper*groupj+i]);
      }
      printf("\n");
  }
  */
}


/*
 * Jacobian routine. COmpute J(t,y) = df/dy.
 * This is done on the GPU.
 */
static int Jacobian(realtype t, N_Vector y, N_Vector fy, SUNMatrix J,
    void *user_data, N_Vector tmp1, N_Vector tmp2, N_Vector tmp3)
{
    abc_data *data = (abc_data*)user_data;

    int nnzper;
    realtype *Jdata, *ydata;
    nnzper = 5* 5;
    ydata = N_VGetDeviceArrayPointer_Cuda(y);
    Jdata = SUNMatrix_cuSparse_Data(J);

    jacobian_kernel<<<GRIDSIZE, BLOCKSIZE>>>(ydata, Jdata, *data);

    hipDeviceSynchronize();
    hipError_t cuerr = hipGetLastError();
    if (cuerr != hipSuccess) {
        fprintf(stderr, ">>> ERROR in Jac: hipGetLastError returned %s\n",
                hipGetErrorName(cuerr));
        return(-1);
    }

  return(0);

}


// now write tests kit

void test_interpolation_kernel(abc_data data)
{
    int NSYSTEM = 1024;
    // initialize temperature;
    for (int i = 0; i < NSYSTEM; i++)
    {
        data.Ts[i] = (double) 3000.0 * (i+10)/NSYSTEM;
        data.logTs[i] = log(data.Ts[i]);
    }

    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for (int j = 0; j < 1; j++){
    linear_interpolation_kernel<<<GRIDSIZE, BLOCKSIZE>>>(data);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time to generate:  %3.1f ms \n", time);
    
    hipDeviceSynchronize();
    hipError_t cuerr = hipGetLastError();
    if (cuerr != hipSuccess) {
        fprintf(stderr,
        ">>> ERROR in interpolation_kernel: hipGetLastError returned %s\n",
        hipGetErrorName(cuerr));
    }
}

void initialize_ydata(double *ydata, int NSYSTEM)
{
    int nchem = 5;
    for (int i = 0; i < NSYSTEM; i++)
    {
        // H2I
        ydata[i*nchem]   = 1.0;
        // H2II
        ydata[i*nchem+1] = 1.0;
        // HI
        ydata[i*nchem+2] = 10.0;
        // HII
        ydata[i*nchem+3] = 1.0;
        // H-
        ydata[i*nchem+4] = 1.0;
    }
}


void test_temperature_kernel(abc_data data)
{
    int NSYSTEM = 1024;
    int nchem   = 5;
    int neq = NSYSTEM*nchem;

    N_Vector y = N_VNew_Cuda(neq);
    double *ydata;
    ydata = N_VGetHostArrayPointer_Cuda(y);
    initialize_ydata(ydata, NSYSTEM);
    N_VCopyToDevice_Cuda(y);


    ydata = N_VGetDeviceArrayPointer_Cuda(y);
    temperature_kernel<<<GRIDSIZE,BLOCKSIZE>>>(ydata, data);

    for (int i = 0; i<NSYSTEM; i++){
        printf("temperature[%d] = %0.5g\n", i, data.Ts[i]);
    }

    hipDeviceSynchronize();
    hipError_t cuerr = hipGetLastError();
    if (cuerr != hipSuccess) {
        fprintf(stderr,
        ">>> ERROR in temperature kernel: hipGetLastError returned %s\n",
        hipGetErrorName(cuerr));
    }
}



void test_rhs_function(abc_data data)
{
    double t = 1.0;
    int NSYSTEM = 1024;
    int nchem   = 5;
    int neq = NSYSTEM*nchem;
    
    N_Vector y = N_VNew_Cuda(neq);
    double *ydata;
    ydata = N_VGetHostArrayPointer_Cuda(y);
    initialize_ydata(ydata, NSYSTEM);
    N_VCopyToDevice_Cuda(y);


    ydata = N_VGetDeviceArrayPointer_Cuda(y);
    N_Vector ydot = N_VNew_Cuda(neq);

    f(t, y, ydot, &data);
    //f(realtype t, N_Vector y, N_Vector ydot, void *user_data)
}


void test_jacobian_function(abc_data data)
{
    double t = 1.0;
    int NSYSTEM = 1024;
    int nchem   = 5;
    int neq = NSYSTEM*nchem;
    
    N_Vector y = N_VNew_Cuda(neq);
    double *ydata;
    ydata = N_VGetHostArrayPointer_Cuda(y);
    initialize_ydata(ydata, NSYSTEM);
    N_VCopyToDevice_Cuda(y);

    ydata = N_VGetDeviceArrayPointer_Cuda(y);
    N_Vector ydot = N_VNew_Cuda(neq);
    
    // also need to initialize jacobian data space

  /* Create sparse SUNMatrix for use in linear solves */
    SUNMatrix A;
    A = NULL;

  hipsparseHandle_t cusp_handle;
  hipsparseCreate(&cusp_handle);
  A = SUNMatrix_cuSparse_NewBlockCSR(NSYSTEM, nchem, nchem, nchem*nchem, cusp_handle);

  /* Initialiize the Jacobian with its fixed sparsity pattern */
  JacInit(A);

    Jacobian(t, y, y, A, &data, y, y, y);
    //f(realtype t, N_Vector y, N_Vector ydot, void *user_data)
}

/*
 * Private Helper Function
 * Get and print some final statistics
 */

/*
 * Check function return value...
 *   opt == 0 means SUNDIALS function allocates memory so check if
 *            returned NULL pointer
 *   opt == 1 means SUNDIALS function returns an integer value so check if
 *            retval < 0
 *   opt == 2 means function allocates memory so check if returned
 *            NULL pointer
 */

static int check_retval(void *returnvalue, const char *funcname, int opt)
{
  int *retval;

  /* Check if SUNDIALS function returned NULL pointer - no memory allocated */
  if (opt == 0 && returnvalue == NULL) {
    fprintf(stderr, "\nSUNDIALS_ERROR: %s() failed - returned NULL pointer\n\n",
	    funcname);
    return(1); }

  /* Check if retval < 0 */
  else if (opt == 1) {
    retval = (int *) returnvalue;
    if (*retval < 0) {
      fprintf(stderr, "\nSUNDIALS_ERROR: %s() failed with retval = %d\n\n",
	      funcname, *retval);
      return(1); }}

  /* Check if function returned NULL pointer - no memory allocated */
  else if (opt == 2 && returnvalue == NULL) {
    fprintf(stderr, "\nMEMORY_ERROR: %s() failed - returned NULL pointer\n\n",
	    funcname);
    return(1); }

  return(0);
}

static void PrintFinalStats(void *cvode_mem, SUNLinearSolver LS)
{
  long int nst, nfe, nsetups, nje, nni, ncfn, netf, nge;
  size_t cuSpInternalSize, cuSpWorkSize;
  int retval;

  retval = CVodeGetNumSteps(cvode_mem, &nst);
  check_retval(&retval, "CVodeGetNumSteps", 1);
  retval = CVodeGetNumRhsEvals(cvode_mem, &nfe);
  check_retval(&retval, "CVodeGetNumRhsEvals", 1);
  retval = CVodeGetNumLinSolvSetups(cvode_mem, &nsetups);
  check_retval(&retval, "CVodeGetNumLinSolvSetups", 1);
  retval = CVodeGetNumErrTestFails(cvode_mem, &netf);
  check_retval(&retval, "CVodeGetNumErrTestFails", 1);
  retval = CVodeGetNumNonlinSolvIters(cvode_mem, &nni);
  check_retval(&retval, "CVodeGetNumNonlinSolvIters", 1);
  retval = CVodeGetNumNonlinSolvConvFails(cvode_mem, &ncfn);
  check_retval(&retval, "CVodeGetNumNonlinSolvConvFails", 1);

  retval = CVodeGetNumJacEvals(cvode_mem, &nje);
  check_retval(&retval, "CVodeGetNumJacEvals", 1);

  retval = CVodeGetNumGEvals(cvode_mem, &nge);
  check_retval(&retval, "CVodeGetNumGEvals", 1);

  SUNLinSol_cuSolverSp_batchQR_GetDeviceSpace(LS, &cuSpInternalSize, &cuSpWorkSize);

  printf("\nFinal Statistics:\n");
  printf("nst = %-6ld nfe  = %-6ld nsetups = %-6ld nje = %ld\n",
	 nst, nfe, nsetups, nje);
  printf("nni = %-6ld ncfn = %-6ld netf = %-6ld    nge = %ld\n \n",
   nni, ncfn, netf, nge);
  printf("cuSolverSp numerical factorization workspace size (in bytes) = %ld\n", cuSpWorkSize);
  printf("cuSolverSp internal Q, R buffer size (in bytes) = %ld\n", cuSpInternalSize);
}


int run_solver(int argc, char *argv[])
{
    realtype reltol, t, tout;
    realtype *ydata, *abstol_data;
    N_Vector y, abstol;
    SUNMatrix A;
    SUNLinearSolver LS;
    void *cvode_mem;
    int retval, iout;
    int neq, ngroups, groupj;
    abc_data data = abc_setup_data(NULL, NULL);
    abc_read_cooling_tables( &data);
    abc_read_rate_tables( &data);

    hipsparseHandle_t cusp_handle;
    hipsolverSpHandle_t cusol_handle;

    y = abstol = NULL;
    A = NULL;
    LS = NULL;
    cvode_mem = NULL;

    /* Parse command line arguments */
    ngroups = BATCHSIZE;
    int GROUPSIZE = 5;
    neq = ngroups* GROUPSIZE;

    int NSYSTEM = BATCHSIZE;
    reltol = 1.0e-5;
    /* Initialize cuSOLVER and cuSPARSE handles */
    hipsparseCreate(&cusp_handle);
    hipsolverSpCreate(&cusol_handle);

    /* Create CUDA vector of length neq for I.C. and abstol */
    y = N_VNew_Cuda(neq);
    if (check_retval((void *)y, "N_VNew_Cuda", 0)) return(1);
    abstol = N_VNew_Cuda(neq);
    if (check_retval((void *)abstol, "N_VNew_Cuda", 0)) return(1);
    
    ydata = N_VGetHostArrayPointer_Cuda(y);
    abstol_data = N_VGetHostArrayPointer_Cuda(abstol);

    /* Initialize */
    initialize_ydata(ydata, NSYSTEM);
    for (int i = 0; i < neq; i++){
        abstol_data[i] = 1.0e-5;
    }
    N_VCopyToDevice_Cuda(y);
    N_VCopyToDevice_Cuda(abstol);

    /* Call CVodeCreate to create the solver memory and specify the
    * Backward Differentiation Formula */
    cvode_mem = CVodeCreate(CV_BDF);
    if (check_retval((void *)cvode_mem, "CVodeCreate", 0)) return(1);

    /* Call CVodeInit to initialize the integrator memory and specify the
    * user's right hand side function in y'=f(t,y), the inital time T0, and
    * the initial dependent variable vector y. */
    retval = CVodeInit(cvode_mem, f, T0, y);
    if (check_retval(&retval, "CVodeInit", 1)) return(1);

    /* Call CVodeSetUserData to attach the user data structure */
    retval = CVodeSetUserData(cvode_mem, &data);
    if (check_retval(&retval, "CVodeSetUserData", 1)) return(1);

    /* Call CVodeSVtolerances to specify the scalar relative tolerance
    * and vector absolute tolerances */
    retval = CVodeSVtolerances(cvode_mem, reltol, abstol);
    if (check_retval(&retval, "CVodeSVtolerances", 1)) return(1);

    /* Create sparse SUNMatrix for use in linear solves */
    A = SUNMatrix_cuSparse_NewBlockCSR(ngroups, GROUPSIZE, GROUPSIZE, GROUPSIZE*GROUPSIZE, cusp_handle);
    if(check_retval((void *)A, "SUNMatrix_cuSparse_NewBlockCSR", 0)) return(1);

    /* Set the sparsity pattern to be fixed so that the row pointers
    * and column indicies are not zeroed out by SUNMatZero */
    retval = SUNMatrix_cuSparse_SetFixedPattern(A, 1);

    /* Initialiize the Jacobian with its fixed sparsity pattern */
    blockJacInit(A);

    /* Create the SUNLinearSolver object for use by CVode */
    LS = SUNLinSol_cuSolverSp_batchQR(y, A, cusol_handle);
    if(check_retval((void *)LS, "SUNLinSol_cuSolverSp_batchQR", 0)) return(1);

    /* Call CVodeSetLinearSolver to attach the matrix and linear solver to CVode */
    retval = CVodeSetLinearSolver(cvode_mem, LS, A);
    if(check_retval(&retval, "CVodeSetLinearSolver", 1)) return(1);

    /* Set the user-supplied Jacobian routine Jac */
    retval = CVodeSetJacFn(cvode_mem, Jacobian);
    if(check_retval(&retval, "CVodeSetJacFn", 1)) return(1);

    /* In loop, call CVode, print results, and test for error.
     Break out of loop when NOUT preset output times have been reached.  */
    printf(" \nGroup of independent 3-species kinetics problems\n\n");
    printf("number of groups = %d\n\n", ngroups);
    
    CVodeSetMaxNumSteps(cvode_mem, 1000);

    iout = 0;  tout = 1.0e0;
    while(1) {
        retval = CVode(cvode_mem, tout, y, &t, CV_NORMAL);
        N_VCopyFromDevice_Cuda(y);
        for (groupj = 0; groupj < ngroups; groupj += 512) {
            printf("group %d: @ t = %0.5g\n", groupj, tout);
            for (int i = 0; i < GROUPSIZE; i++){
                printf("ydata[%d] = %0.5g\n", GROUPSIZE*groupj+i, ydata[GROUPSIZE*groupj+i]);
            }
            printf("\n");
        }
        
        if (check_retval(&retval, "CVode", 1)) break;
        if (retval == CV_SUCCESS) {
            iout++;
            tout *= TMULT;
        }

        if (iout == NOUT) break;
    }

  /* Print some final statistics */
  PrintFinalStats(cvode_mem, LS);

  /* Free y and abstol vectors */
  N_VDestroy(y);
  N_VDestroy(abstol);

  /* Free integrator memory */
  CVodeFree(&cvode_mem);

  /* Free the linear solver memory */
  SUNLinSolFree(LS);

  /* Free the matrix memory */
  SUNMatDestroy(A);

  /* Destroy the cuSOLVER and cuSPARSE handles */
  hipsparseDestroy(cusp_handle);
  hipsolverSpDestroy(cusol_handle);

  return(0);
}



int main()
{
    // read the rate data 

    hipDeviceSynchronize();
    abc_data data = abc_setup_data(NULL, NULL);
    abc_read_cooling_tables( &data);
    abc_read_rate_tables( &data);

    // printf("rk01 = %0.5g\n", data.r_k22[213]);
    // printf("h2mheat = %0.5g\n", data.c_h2formation_h2mheat[1020]);  
    
    /*
    // test interpolation first
    test_interpolation_kernel(data);

    // test temperature kerenel
    test_temperature_kernel(data);

    // test the rhs function
    test_rhs_function(data);

    // initialize initial conditions
    // create a y_vec that holds NSYSTEM  * nchem elements
    test_jacobian_function(data);
*/
    // initialize yvec to see if we can have it print out accurate ydot

    run_solver(NULL, NULL);
    hipDeviceSynchronize();
}


int check_vector(N_Vector X, N_Vector Y, realtype tol)
{
  int failure = 0;
  sunindextype i, local_length, maxloc;
  realtype *Xdata, *Ydata, maxerr;

  hipDeviceSynchronize();

  N_VCopyFromDevice_Cuda(X);
  N_VCopyFromDevice_Cuda(Y);

  Xdata = N_VGetHostArrayPointer_Cuda(X);
  Ydata = N_VGetHostArrayPointer_Cuda(Y);
  local_length = N_VGetLength(X);

  /* check vector data */
  for(i=0; i < local_length; i++)
    failure += FNEQ(Xdata[i], Ydata[i], tol);

  if (failure > ZERO) {
    maxerr = ZERO;
    maxloc = -1;
    for(i=0; i < local_length; i++) {
      if (SUNRabs(Xdata[i]-Ydata[i]) >  maxerr) {
        maxerr = SUNRabs(Xdata[i]-Ydata[i]);
        maxloc = i;
      }
    }
    printf("check err failure: maxerr = %g at loc %li (tol = %g)\n",
	   maxerr, (long int) maxloc, tol);
    return(1);
  }
  else
    return(0);
}
void sync_device()
{
  hipDeviceSynchronize();
}
